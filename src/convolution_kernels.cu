#include "hip/hip_runtime.h"
#include "channel.h"

static dim3 threadsPerBlock;
static dim3 blocksPerGrid;


static __global__ void calcOmegakernel(float2* wx,float2* wy,float2* wz,float2* ux,float2* uy,float2* uz,int IGLOBAL)
{
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	int j=k%NY;
	k=(k-j)/NY;

	// [i,k,j][NX,NZ,NY]	

	int h=i*NY*NZ+k*NY+j;


	if (i<NXSIZE && j<NY && k<NZ)
	{
	
	float k1;
	float k3;

	// X indices		
	k1=(i+IGLOBAL)<NX/2 ? (float)(i+IGLOBAL) : (float)(i+IGLOBAL)-(float)NX ;
	
	// Z indices
	k3=(float)k;	

	//Set to LX and LZ
	//Fraction
	k1=(PI2/LX)*k1;
	k3=(PI2/LZ)*k3;	

	float2 u1=ux[h];
	float2 u2=uy[h];
	float2 u3=uz[h];

	float2 w1=wx[h];
	float2 w3=wz[h];
	float2 w2;

	w1.x=w1.x -(-k3*u2.y);
	w1.y=w1.y -k3*u2.x ;
	
	w2.x=-(k3*u1.y-k1*u3.y);
	w2.y=  k3*u1.x-k1*u3.x ;
	
	w3.x=-w3.x-(k1*u2.y);
	w3.y=-w3.y+ k1*u2.x ;		
		
	//Write
	
	wx[h]=w1;
	wy[h]=w2;
	wz[h]=w3;
	
	
	
	}
	
	
}


static __global__ void rotorkernel(float2* wx,float2* wy,float2* wz,float2* ux,float2* uy,float2* uz,int IGLOBAL)
{
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	int j=k%NY;
	k=(k-j)/NY;

	// [i,k,j][NX,NZ,NY]	

	int h=i*NY*NZ+k*NY+j;


	if (i<NXSIZE && j<NY && k<NZ)
	{

	float2 m1;
	float2 m2;
	float2 m3;
	
	
	//Normalisation

	int N2=NX*(2*NZ-2);

	// Read velocity and vorticity	
	
	float2 u1=ux[h];
	float2 u2=uy[h];
	float2 u3=uz[h];
	
	float2 w1=wx[h];
	float2 w2=wy[h];
	float2 w3=wz[h];
	
	// Normalize velocity and vorticity
	
	u1.x=u1.x/N2;
	u2.x=u2.x/N2;
	u3.x=u3.x/N2;

	u1.y=u1.y/N2;
	u2.y=u2.y/N2;
	u3.y=u3.y/N2;

	w1.x=w1.x/N2;
	w2.x=w2.x/N2;
	w3.x=w3.x/N2;
		
	w1.y=w1.y/N2;
	w2.y=w2.y/N2;
	w3.y=w3.y/N2;
	
	// Calculate the convolution rotor

	m1.x=u2.x*w3.x-u3.x*w2.x;
	m2.x=u3.x*w1.x-u1.x*w3.x;
	m3.x=u1.x*w2.x-u2.x*w1.x;

	m1.y=u2.y*w3.y-u3.y*w2.y;
	m2.y=u3.y*w1.y-u1.y*w3.y;
	m3.y=u1.y*w2.y-u2.y*w1.y;

	// Output must be normalized with N^3	
	
	wx[h].x=m1.x;
	wx[h].y=m1.y;

	wy[h].x=m2.x;
	wy[h].y=m2.y;

	wz[h].x=m3.x;
	wz[h].y=m3.y;	


	}
	
	
}

///////////////////FUNCTIONS///////////////////////

extern void calcOmega(float2* wx,float2* wy,float2* wz,float2* ux,float2* uy,float2* uz, domain_t domain){

	
	threadsPerBlock.x=THREADSPERBLOCK_IN;
	threadsPerBlock.y=THREADSPERBLOCK_IN;


	blocksPerGrid.x=NXSIZE/threadsPerBlock.x;
	blocksPerGrid.y=NZ*NY/threadsPerBlock.y;


	calcOmegakernel<<<blocksPerGrid,threadsPerBlock>>>(wx,wy,wz,ux,uy,uz,domain.iglobal);
	kernelCheck(RET,domain,"Boundary");


}

extern void calcRotor(float2* wx,float2* wy,float2* wz,float2* ux,float2* uy,float2* uz, domain_t domain){

	
	threadsPerBlock.x=THREADSPERBLOCK_IN;
	threadsPerBlock.y=THREADSPERBLOCK_IN;


	blocksPerGrid.x=NXSIZE/threadsPerBlock.x;
	blocksPerGrid.y=NZ*NY/threadsPerBlock.y;

	rotorkernel<<<blocksPerGrid,threadsPerBlock>>>(wx,wy,wz,ux,uy,uz,domain.iglobal);
	kernelCheck(RET,domain,"Boundary");


}



