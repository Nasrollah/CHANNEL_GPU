#include"channel.h"

void kernelCheck( hipError_t error, const char* function)
{


	error= hipGetLastError();			
		if(error !=hipSuccess)
		{
			const char* error_string= hipGetErrorString(error);
			printf("\n error  %s : %s RANK=%d \n", function, error_string,RANK);
			exit(1);
		}

	return;
}

extern void cufftCheck( hipfftResult error, const char* function )
{
	if(error != HIPFFT_SUCCESS)
	{
		printf("\n error  %s : %d RANK=%d \n", function, error,RANK);
		exit(1);
	}
		
	return;
}  

extern void cusparseCheck( hipsparseStatus_t error, const char* function )
{
	if(error != HIPSPARSE_STATUS_SUCCESS)
	{
		printf("\n error  %s : %d RANK=%d \n", function, error,RANK);
		exit(1);
	}
		
	return;
}  

extern void cublasCheck(hipblasStatus_t error, const char* function )
{
	if(error !=  HIPBLAS_STATUS_SUCCESS)
	{
		printf("\n error  %s : %d RANK=%d \n", function, error,RANK);
		exit(1);
	}
		
	return;
}  


extern void cudaCheck( hipError_t error, const char* function)
{
	if(error !=hipSuccess)
	{
		const char* error_string= hipGetErrorString(error);
		printf("\n error  %s : %s RANK=%d \n", function, error_string,RANK);
		exit(1);
	}
		

	return;
}



extern void mpiCheck( int error, const char* function)
{
	if(error !=0)
	{
		//printf("\n error_MPI %s \n",(char*)function);
		printf("error_mpi");		
		exit(1);
	}
		
	

	return;
}



