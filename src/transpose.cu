#include "hip/hip_runtime.h"

#include"channel.h"

static hipblasHandle_t cublasHandle;
static float2 alpha[1];
static float2 betha[1];
static float2* B;


void setTranspose(void){


	cublasCheck(hipblasCreate(&cublasHandle),"Cre");

	alpha[0].x=1.0f;
	alpha[0].y=0.0f;
		

	return;
}

void transpose_A(float2* u_2,float2* u_1){

	//Transpuesta de [i,k,j][NX,NZ,NY] a -----> [j,i,k][NY,NX,NZ]

	cublasCheck(hipblasCgeam(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_T,NX*NZ,NY,alpha,(const float2*)u_1,NY,0,0,NY,u_2,NX*NZ),"Tr");
	//printf("\n%f,%f",alpha[0].x,alpha[0].y);
	return;


}

void transpose_B(float2* u_2,float2* u_1){

	//Transpuesta de [j,i,k][NY,NX,NZ] a -----> [i,k,j][NX,NZ,NY]

	cublasCheck(hipblasCgeam(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_T,NY,NX*NZ,alpha,(const float2*)u_1,NX*NZ,0,0,NX*NZ,u_2,NY),"Tr");
	//printf("\n%f,%f",alpha[0].x,alpha[0].y);
	return;

}
