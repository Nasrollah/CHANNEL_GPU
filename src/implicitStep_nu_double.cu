#include "hip/hip_runtime.h"
#include"channel.h"

static __global__ void cast_kernel(float2* u,double2* v)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	int j=k%NY;
	k=(k-j)/NY;

	int h=i*NY*NZ+k*NY+j;

	if(i<NXSIZE/NSTEPS & k<NZ & j<NY){

	float2 ud;
	double2 vd;

	vd=v[h];

	ud.x=__double2float_rn(vd.x);
	ud.y=__double2float_rn(vd.y);


	u[h]=ud;

	}

}


static __global__ void rhs_A_kernel(double2* v,float2* u)
{  

		//Define shared memory


		__shared__ double2 sf[NY+2];

		int k   = blockIdx.x;
		int i   = blockIdx.y;

		int j   = threadIdx.x;


		int h=i*NZ*NY+k*NY+j;

		double2 u_temp;
		
		double2 ap_1;
		double2 ac_1;
		double2 am_1;

		double a=Fmesh((j+1)*DELTA_Y-1.0)-Fmesh(j*DELTA_Y-1.0);
		double b=Fmesh((j-1)*DELTA_Y-1.0)-Fmesh(j*DELTA_Y-1.0);	

		double alpha=-(-b*b*b-a*b*b+a*a*b)/(a*a*a-4.0*a*a*b+4.0*a*b*b-b*b*b);
		double betha=-( a*a*a+b*a*a-b*b*a)/(a*a*a-4.0*a*a*b+4.0*a*b*b-b*b*b);


		if(i<NXSIZE/NSTEPS & k<NZ & j<NY){

		//Read from global so shared

		sf[j+1].x=(double)u[h].x;
		sf[j+1].y=(double)u[h].y;

		__syncthreads();

		ap_1=sf[j+2];	
		ac_1=sf[j+1];
		am_1=sf[j];
		
		
		u_temp.x=(alpha*ap_1.x+ac_1.x+betha*am_1.x);
		u_temp.y=(alpha*ap_1.y+ac_1.y+betha*am_1.y);

		if(j==0){
		u_temp.x=0.0;
		u_temp.y=0.0;		
		}		
	
		if(j==NY-1){
		u_temp.x=0.0;
		u_temp.y=0.0;		
		}	

		v[h]=u_temp;
 	
	  }

}

static __global__ void setDiagkernel(double2* ldiag,double2* cdiag,double2* udiag,float bethaDt,int nstep,int IGLOBAL){  

		
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		int k = blockIdx.y * blockDim.y + threadIdx.y;

		int j=k%NY;
		k=(k-j)/NY;

		// [i,k,j][NX,NZ,NY]	

		int h=i*NY*NZ+k*NY+j;

		if (i<NXSIZE/NSTEPS && j<NY && k<NZ)
		{

		double k1;
		double k3;
	
		double kk;
	
		int stride=nstep*NXSIZE/NSTEPS;	

		// X indices		
		k1=(i+IGLOBAL+stride)<NX/2 ? (double)(i+IGLOBAL+stride) : (double)(i+IGLOBAL+stride)-(double)NX ;
		
		// Z indices
		k3=(double)k;
	
		//Fraction
		k1=(PI2/LX)*k1;
		k3=(PI2/LZ)*k3;	

		kk=k1*k1+k3*k3;

		double nu=1.0/REYNOLDS;
		double D=nu*bethaDt;

		//COEFICIENTS OF THE NON_UNIFORM GRID

		double a=Fmesh((j+1)*DELTA_Y-1.0)-Fmesh(j*DELTA_Y-1.0);
		double b=Fmesh((j-1)*DELTA_Y-1.0)-Fmesh(j*DELTA_Y-1.0);	
	

		double A=-12.0*b/(a*a*a-b*b*b-4.0*a*a*b+4.0*b*b*a);
		double B= 12.0*a/(a*a*a-b*b*b-4.0*a*a*b+4.0*b*b*a);
		double C=-A-B;
		
		double alpha=-(-b*b*b-a*b*b+a*a*b)/(a*a*a-4.0*a*a*b+4.0*a*b*b-b*b*b);
		double betha=-( a*a*a+b*a*a-b*b*a)/(a*a*a-4.0*a*a*b+4.0*a*b*b-b*b*b);


		//veamos
	
		double2 ldiag_h;
		double2 cdiag_h;
		double2 udiag_h;
	
		ldiag_h.x=betha-D*B+D*kk*betha;
		ldiag_h.y=0.0;			
	
		cdiag_h.x=1.0-D*C+D*kk*1.0;
		cdiag_h.y=0.0;		
	
		udiag_h.x=alpha-D*A+D*kk*alpha;
		udiag_h.y=0.0;	

		//To be improved 
		
		if(j==0){
		ldiag_h.x=0.0;
		cdiag_h.x=1.0;
		udiag_h.x=0.0;
		}
	
		if(j==1){
		ldiag_h.x=0.0;
		}

		if(j==NY-1){
		ldiag_h.x=0.0;
		cdiag_h.x=1.0;
		udiag_h.x=0.0;
		}	
	
		if(j==NY-2){
		udiag_h.x=0.0;
		}

		// Write		

		ldiag[h]=ldiag_h;
		cdiag[h]=cdiag_h;
		udiag[h]=udiag_h;			
	
	  }

}



static dim3 threadsPerBlock;
static dim3 blocksPerGrid;


static dim3 threadsPerBlock_B;
static dim3 blocksPerGrid_B;

static hipsparseHandle_t implicit_handle;

extern void setImplicitDouble(void){

	threadsPerBlock.x=NY;
	threadsPerBlock.y=1;

	blocksPerGrid.x=NZ;
	blocksPerGrid.y=NXSIZE/NSTEPS;	

	threadsPerBlock_B.x= THREADSPERBLOCK_IN;
	threadsPerBlock_B.y= THREADSPERBLOCK_IN;

	blocksPerGrid_B.x=NXSIZE/NSTEPS/threadsPerBlock_B.x;
	blocksPerGrid_B.y=NZ*NY/threadsPerBlock_B.y;


	cusparseCheck(hipsparseCreate(&implicit_handle),"Handle");

}

extern void implicitSolver_double(float2* u,float betha,float dt){

	
	//SIZE OF LDIAG CDIAG UDIAG AND AUX
	//2*SIZE/NSTEPS

	for(int i=0;i<NSTEPS;i++){

	setDiagkernel<<<blocksPerGrid_B,threadsPerBlock_B>>>(LDIAG,CDIAG,UDIAG,dt*betha,i,IGLOBAL);

	rhs_A_kernel<<<blocksPerGrid,threadsPerBlock>>>(AUX,u+i*NXSIZE/NSTEPS*NZ*NY);
	kernelCheck(RET,"hemholz");	

	cusparseCheck(cusparseZgtsvStridedBatch(implicit_handle,NY,LDIAG,CDIAG,UDIAG,AUX,NXSIZE/NSTEPS*NZ,NY),"HEM");

	cast_kernel<<<blocksPerGrid_B,threadsPerBlock_B>>>(u+i*NXSIZE/NSTEPS*NZ*NY,AUX);

	}
	return;

}






